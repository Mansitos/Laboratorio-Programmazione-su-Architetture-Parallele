// file di esempio/test compilazione


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void my_kernel( void ) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    printf( "Sono il thread %d! block %d!\n", tid , blockIdx.x);
}

int main( void ) {  
    my_kernel<<<2,32>>>();  // 2 blocchi da 32 thread ciascuno
    printf( "Hello, World!\n" );
    cout << "Inserire qualsiasi carattere per interrompere il processo\n";
    string x;
    cin >> x;
    return 0;
}