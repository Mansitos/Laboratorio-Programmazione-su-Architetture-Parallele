
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

const int N = 1 << 20;

__global__ void kernel(float *x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
        for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
            x[i] = sqrt(pow(3.14159,i));
    }
}

int main(){
    const int num_streams = 8;
    hipStream_t streams[num_streams]; // array di streams
    float *data[num_streams];   // (matrice di float??)

    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]); // crea uno degli 8 stream
        hipMalloc(&data[i], N * sizeof(float));
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N); // lancia un kernel per stream
        kernel<<<1, 1>>>(0, 0); // lancia un “dummy kernel” nel default stream
    }
    hipDeviceReset();
    return 0;
}